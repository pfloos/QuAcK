
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>


__global__ void elementwise_dsqrt_inplace_kernel(int nS, double *A) {


    int i;

    i = blockIdx.x * blockDim.x + threadIdx.x;

    while(i < nS) {

        if(A[i] > 0.0) {

            A[i] = sqrt(A[i]);

        } else {

            A[i] = sqrt(-A[i]);

        }

        i += blockDim.x * gridDim.x;
    } // i

}





extern "C" void elementwise_dsqrt_inplace(int nS, double *A) {

    int sBlocks = 32;
    int nBlocks = (nS + sBlocks - 1) / sBlocks;

    dim3 dimGrid(nBlocks, 1, 1);
    dim3 dimBlock(sBlocks, 1, 1);

    printf("lunching elementwise_dsqrt_inplace_kernel with %d blocks and %d threads/block\n",
        nBlocks, sBlocks);


    elementwise_dsqrt_inplace_kernel<<<dimGrid, dimBlock>>>(nS, A);

}




